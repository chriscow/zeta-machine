#include "hip/hip_runtime.h"
#include <math.h>
#include <iostream>
#include <hip/hip_math_constants.h>
#include <hip/hip_runtime.h>
#include "zeta.hpp"
#include "bitmap.hpp"

__device__
int cc[300] = { 0, 0, 0, 0, 0, 0, 0, 0, 255, 0, 60, 255, 0, 100, 255, 0, 125, 255, 0, 140, 255, 0, 155, 255, 0, 170, 255, 0, 180, 255, 0, 190, 255, 0, 200, 255, 0, 210, 255, 0, 220, 255, 0, 225, 255, 0, 230, 255, 0, 235, 255, 0, 238, 255, 0, 241, 255, 0, 244, 255, 0, 247, 255, 0, 250, 255, 0, 253, 255, 0, 255, 255, 0, 255, 255, 0, 255, 255, 0, 255, 255, 0, 255, 255, 0, 255, 255, 0, 255, 255, 0, 255, 255, 0, 255, 255, 0, 255, 255, 0, 255, 0, 0, 255, 0, 0, 255, 0, 0, 255, 0, 0, 255, 0, 0, 255, 0, 0, 255, 0, 0, 255, 0, 0, 255, 0, 0, 255, 0, 23, 255, 0, 46, 255, 0, 70, 255, 0, 93, 255, 0, 116, 255, 0, 139, 255, 0, 163, 255, 0, 186, 255, 0, 209, 255, 0, 232, 255, 0, 255, 255, 0, 255, 237, 0, 255, 218, 0, 255, 200, 0, 255, 181, 0, 255, 163, 0, 255, 146, 0, 255, 128, 0, 255, 111, 0, 255, 93, 0, 255, 76, 0, 255, 63, 0, 255, 51, 0, 255, 42, 0, 255, 39, 0, 255, 36, 0, 255, 33, 0, 255, 30, 0, 255, 27, 0, 255, 24, 0, 255, 21, 0, 255, 18, 0, 255, 15, 0, 255, 12, 0, 255, 9, 0, 255, 6, 0, 255, 3, 0, 255, 0, 0, 255, 0, 0, 255, 0, 0, 255, 0, 0, 255, 0, 0, 255, 0, 0, 255, 0, 0, 255, 0, 0, 255, 0, 0, 255, 0, 0, 255, 0, 0, 255, 0, 0, 255, 0, 0, 255, 0, 0, 255, 0, 0, 255, 0, 0, 255, 0, 0, 255, 0, 0, 255, 0, 0, 255, 0, 0 };

int cc_[300] = { 0, 0, 0, 0, 0, 0, 0, 0, 255, 0, 60, 255, 0, 100, 255, 0, 125, 255, 0, 140, 255, 0, 155, 255, 0, 170, 255, 0, 180, 255, 0, 190, 255, 0, 200, 255, 0, 210, 255, 0, 220, 255, 0, 225, 255, 0, 230, 255, 0, 235, 255, 0, 238, 255, 0, 241, 255, 0, 244, 255, 0, 247, 255, 0, 250, 255, 0, 253, 255, 0, 255, 255, 0, 255, 255, 0, 255, 255, 0, 255, 255, 0, 255, 255, 0, 255, 255, 0, 255, 255, 0, 255, 255, 0, 255, 255, 0, 255, 255, 0, 255, 0, 0, 255, 0, 0, 255, 0, 0, 255, 0, 0, 255, 0, 0, 255, 0, 0, 255, 0, 0, 255, 0, 0, 255, 0, 0, 255, 0, 23, 255, 0, 46, 255, 0, 70, 255, 0, 93, 255, 0, 116, 255, 0, 139, 255, 0, 163, 255, 0, 186, 255, 0, 209, 255, 0, 232, 255, 0, 255, 255, 0, 255, 237, 0, 255, 218, 0, 255, 200, 0, 255, 181, 0, 255, 163, 0, 255, 146, 0, 255, 128, 0, 255, 111, 0, 255, 93, 0, 255, 76, 0, 255, 63, 0, 255, 51, 0, 255, 42, 0, 255, 39, 0, 255, 36, 0, 255, 33, 0, 255, 30, 0, 255, 27, 0, 255, 24, 0, 255, 21, 0, 255, 18, 0, 255, 15, 0, 255, 12, 0, 255, 9, 0, 255, 6, 0, 255, 3, 0, 255, 0, 0, 255, 0, 0, 255, 0, 0, 255, 0, 0, 255, 0, 0, 255, 0, 0, 255, 0, 0, 255, 0, 0, 255, 0, 0, 255, 0, 0, 255, 0, 0, 255, 0, 0, 255, 0, 0, 255, 0, 0, 255, 0, 0, 255, 0, 0, 255, 0, 0, 255, 0, 0, 255, 0, 0, 255, 0, 0 };


__device__
real b_coeff[20] = {
    1.0000000000000000000000000000000,
    0.0833333333333333333333333333333,
    -0.0013888888888888888888888888888,
    3.3068783068783068783068783068783E-5,
    -8.2671957671957671957671957671958E-7,
    2.0876756987868098979210090321201E-8,
    -5.2841901386874931848476822021796E-10,
    1.3382536530684678832826980975129E-11,
    -3.3896802963225828668301953912494E-13,
    8.5860620562778445641359054504256E-15,
    -2.1748686985580618730415164238659E-16,
    5.5090028283602295152026526089023E-18,
    -1.3954464685812523340707686264064E-19,
    3.5347070396294674716932299778038E-21,
    -8.9535174266605480875210207537274E-23,
    2.2679524523376830603109507388682E-24
    -5.7447906688722024452638819876070E-26,
    1.4551724756148649018662648672713E-27,
    -3.6859949406653101781817824799086E-29,
    9.3367342570950446720325551527856E-31
};

__device__
real g_coeff[15] = {
    0.99999999999999709182,
    57.15623566586292351700,
    -59.59796035547549124800,
    14.13609797474174717400,
    -0.491913816097620199780,
    0.33994649984811888699E-4,
    0.46523628927048575665E-4,
    -0.98374475304879564677E-4,
    0.15808870322491248884E-3,
    -0.21026444172410488319E-3,
    0.21743961811521264320E-3,
    -0.16431810653676389022E-3,
    0.84418223983852743293E-4,
    -0.26190838401581408670E-4,
    0.36899182659531622704E-5
};

std::ostream& operator<<(std::ostream &out, const complex &z) {
    out << z.re << " + " << z.im << "i";
    return out;
}
std::ostream& operator<<(std::ostream &out, const Settings &settings) {
    out << "[" << settings.min << ", " << settings.max << "], " << settings.res << "ppu";
    return out;
}

__device__
void print_complex(const char* label, int i, complex z) {
    printf("%s %d %.16lG + %.16lGi\n", label, i, z.re, z.im);
}

__device__
real complex::arg() const {
    if (im != 0.0) return 2.0 * atan((mod() - re) / im);
    if (re > 0.0) return 0.0;
    if (re < 0.0) return HIP_PI;
    return HIP_NAN;
}

__device__
real complex::mod() const {
    return sqrt((re * re) + (im * im));
}

__device__
complex complex::ccos(complex const& c) {
    return complex(
        cos(c.re) * cosh(c.im),
        -1.0 * sin(c.re) * sinh(c.im)
    );
}

__device__
complex complex::cexp(complex const& c) {
    return complex(
        exp(c.re) * cos(c.im),
        exp(c.re) * sin(c.im)
    );
}

__device__
complex complex::clog(complex const& c) {
    return complex(
        log(c.mod()),
        c.arg()
    );
}

__device__
complex complex::cpow(real x, complex const& exp) {
    return complex(
        pow(x, exp.re) * cos((exp.im) * log(x)),
        pow(x, exp.re) * sin((exp.im) * log(x))
    );
}

__device__
complex complex::cpow(complex const& x, complex const& exp) {
    return cexp(exp * clog(x));
}

__device__
complex zeta(complex s) {
    complex z = 0, g = 1;
    if (s.re < 0) {
        if (abs(s.im) < MAX_GAMMA) {
            s = complex(1) - s;
            g = complex_gamma(s);
            z = ems(s);
            z = z * g * complex(2);
            z = z * complex::cpow(TWO_PI, -s);
            z = z * complex::ccos(complex(HIP_PIO2) * s);
        }
        else {
            z = ems(s);
        }
    }
    else {
        z = ems(s);
    }
    return z;
}

__device__
complex ems(complex s) {
    int N = (int) s.mod(), k;
    complex z = 0, t = 0, temp = 0;
    if (N > MAX_N) N = MAX_N;
    if (N < MIN_N) N = MIN_N;
    for (k = 1; k < N; k++) {
        z = z + complex::cpow(k, -s);
    }
    // print_complex("ems", 0, z);
    z = z + (complex::cpow(N, complex(1) - s) / (s - 1));
    // print_complex("ems", 1, z);
    // z = fp_add(z, fp_div(fp_i_c_pow(N, fp_add(complex(1.0, 0.0), fp_neg(s))), fp_add(s, complex(-1.0, 0.0))));
    z = z + complex(0.5) * complex::cpow(N, -s);
    // print_complex("ems", 2, z);
    // z = fp_add(z, fp_mul(complex(0.5, 0.0), fp_i_c_pow(N, fp_neg(s))));
    for (k = 1; k < 20; k++) {
        t = t + complex(b_coeff[k]) * pochhammer(s, (2 * k) - 1) * complex::cpow(N, complex(1) - s - (2 * k));
        // t = fp_add(t, fp_mul(complex(b_coeff[k], 0.0), fp_mul(fp_pochhammer(s, (2 * k) - 1), fp_i_c_pow(N, fp_add(complex(1 - (2 * k), 0.0), fp_neg(s))))));
        if ((t - temp).re == 0) {
            break;
        }
        // if (fp_add(t, fp_neg(temp)).re == 0.0) break;
        temp = t;
    }
    // print_complex("ems", 3, z);
    return z + t;
}

__device__
complex pochhammer(complex s, int n) {
    int i;
    complex poch_val = 1;
    for (i = 0; i < n; i++) {
        poch_val = poch_val * (s + i);
        // poch_val = fp_mul(poch_val, fp_add(s, complex(i, 0.0)));
    }
    return poch_val;
}

__device__
complex complex_gamma(complex s) {
    int i;
    complex g = g_coeff[0];
    s = s - 1;
    for (i = 1; i < 15; i++) {
        g = g + (complex(g_coeff[i]) / (s + i));
        // g = fp_add(g, fp_div(complex(g_coeff[i], 0.0), fp_add(s, complex(i, 0.0))));
    }
    g = g * complex(SQRT_TWO_PI);
    // g = fp_mul(g, complex(SQRT_TWO_PI, 0.0));
    g = g * complex::cpow(s + complex(5.2421875), s + complex(0.5));
    // g = fp_mul(g, fp_c_c_pow(fp_add(s, complex(5.2421875, 0.0)), fp_add(s, complex(0.5, 0.0))));
    g = g * complex::cexp(complex(-5.2421875) - s);
    // g = fp_mul(g, fp_exp(fp_add(complex(-5.2421875, 0.0), fp_neg(s))));
    return g;
}

__device__
int iterate(complex s) {
    int i = 0;
    real cabs_z = 0.0, diff = 100;
    complex z = 0;
    while (!isnan(cabs_z) && diff > EPSILON && cabs_z < CABS_Z_MAX && i < MAX_ITS) {
        z = zeta(s);
        // print_complex("iter", i, z);
        diff = (z - s).mod();
        // diff = abs(z.re - s.re);
        cabs_z = z.mod();
        i++;
        s = z;
    }
    if (!isnan(cabs_z)) {
        if (cabs_z >= CABS_Z_MAX) {
            if (z.re < 0.0) {
                i += 1;
            } else {
                i += 1;
                i += 1;
            }
        }
    }
    return i;
}

__global__
void zeta_kernel(Settings settings, int width, int height, byte *pixels) {
    int start = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = start; i < width * height; i += stride) {
        int x = i % width;
        int y = i / width;
        real u = x / (real) width;
        real v = y / (real) height;
        complex min = settings.min;
        complex max = settings.max;
        complex range = max - min;
        complex s = min + complex(range.re * u, range.im * v);
        int iterations = iterate(s);
        pixels[(i * 3) + 0] = cc[(iterations * 3) + 0];
        pixels[(i * 3) + 1] = cc[(iterations * 3) + 1];
        pixels[(i * 3) + 2] = cc[(iterations * 3) + 2];
        // pixels[(i * 3) + 0] = u * 255;
        // pixels[(i * 3) + 1] = v * 255;
        // pixels[(i * 3) + 2] = 128;
    }
}

__global__
void calc_iterations(complex min, complex max, uint size, uint *data) {
    int start = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = start; i < size*size; i += stride) {
        int x = i % size;
        int y = i / size;
        real u = x / (real) size;
        real v = y / (real) size;
        complex range = max - min;
        complex s = min + complex(range.re * u, range.im * v);
        int iterations = iterate(s);
        data[i] = iterations;
    }
}

__global__
void zeta_kernel_lut(Settings settings, int width, int height, byte *pixels, ZetaLUTCollection *luts) {
    int start = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = start; i < width * height; i += stride) {
        int x = i % width;
        int y = i / width;
        real u = x / (real) width;
        real v = y / (real) height;
        complex min = settings.min;
        complex max = settings.max;
        complex range = max - min;
        complex s = min + complex(range.re * u, range.im * v);
        rgb_t color = {0, 0, 0};
        if (!luts->lookup(zeta(s), &color)) {
            color = {0, 0, 0};
        }
        pixels[(i * 3) + 0] = color.red;
        pixels[(i * 3) + 1] = color.green;
        pixels[(i * 3) + 2] = color.blue;
    }
}

__global__
void zeta_kernel_single(complex z) {
    iterate(z);
}

__host__
void write_image(Settings settings) {
    int width = (int) ((settings.max.re - settings.min.re) * settings.res);
    int height = (int) ((settings.max.im - settings.min.im) * settings.res);
    int procCount;
    int devId;
    hipGetDevice(&devId);
    hipDeviceGetAttribute(&procCount, hipDeviceAttributeMultiprocessorCount, devId);
    byte *pixels;
    hipMallocManaged(&pixels, width * height * 3);
    //zeta_kernel<<<1, 1>>>(settings, width, height, pixels);
    zeta_kernel<<<procCount * 32, 256>>>(settings, width, height, pixels);
    hipDeviceSynchronize();
    bitmap_image image(width, height);
    image.clear();
    for (int i = 0; i < width; i ++) {
        for (int j = 0; j < height; j ++) {
            int pixel = (j * width + i) * 3;
            rgb_t color = {
                pixels[pixel + 0],
                pixels[pixel + 1],
                pixels[pixel + 2]
            };
            image.set_pixel(i, height - j - 1, color);
        }
    }
    hipFree(pixels);
    image.save_image("zeta.bmp");
}

__host__
void write_image_(int width, int height, uint* data) {
    unsigned char iterations;
    bitmap_image image(width, height);
    image.clear();
    for (int i = 0; i < width; i ++) {
        for (int j = 0; j < height; j ++) {
            int pixel = (j * width + i);

            iterations = (unsigned char)data[pixel];

            rgb_t color = {
                (unsigned char)cc_[(iterations * 3) + 0],
                (unsigned char)cc_[(iterations * 3) + 1],
                (unsigned char)cc_[(iterations * 3) + 2]
            };

            image.set_pixel(i, height - j - 1, color);
        }
    }
    image.save_image("zeta.bmp");
}

__host__
void write_image_lut(Settings settings, const ZetaLUTCollection& luts) {
    int width = (int) ((settings.max.re - settings.min.re) * settings.res);
    int height = (int) ((settings.max.im - settings.min.im) * settings.res);
    int procCount;
    int devId;
    hipGetDevice(&devId);
    hipDeviceGetAttribute(&procCount, hipDeviceAttributeMultiprocessorCount, devId);
    byte *pixels;
    hipMallocManaged(&pixels, width * height * 3);
    ZetaLUTCollection *sharedluts;
    hipMallocManaged(&sharedluts, sizeof(ZetaLUTCollection));
    *sharedluts = luts;
    //zeta_kernel<<<1, 1>>>(settings, width, height, pixels);
    zeta_kernel_lut<<<procCount * 32, 256>>>(settings, width, height, pixels, sharedluts);
    hipDeviceSynchronize();
    bitmap_image image(width, height);
    image.clear();
    for (int i = 0; i < width; i ++) {
        for (int j = 0; j < height; j ++) {
            int pixel = (j * width + i) * 3;
            rgb_t color = {
                pixels[pixel + 0],
                pixels[pixel + 1],
                pixels[pixel + 2]
            };
            image.set_pixel(i, height - j - 1, color);
        }
    }
    hipFree(pixels);
    hipFree(sharedluts);
    image.save_image("zeta.bmp");
}

__host__
void iterate_single(complex z) {
    zeta_kernel_single<<<1, 1>>>(z);
    hipDeviceSynchronize();
}

// void fp_image(real rl, real rh, real il, real ih, int res)
// {
//     char f_name[256];
//     unsigned char r = 0, g = 0, b = 0;
//     int its, row, pixel, width = ((rh - rl) * res) + 1, height = ((ih - il) * res) + 1, row_bytes = get_row_bytes(width);
//     real EPSILON = 1.0 / pow(10, 15), re, im, d = 1.0 / (real)res;
//     if (rh <= rl || ih <= il || res < 1) return;
//     sprintf(f_name, "RZ %.12lG, %.12lG, %.12lG, %.12lG, %d.bmp", rl, rh, il, ih, res);
//     printf("Initialising bitmap image...");
//     FILE *bmp1 = get_bmp(f_name, width, height, row_bytes, 0);
//     if (!bmp1) return;
//     printf("done\n\n");
//     im = il;
//     for (row = 1; row <= height; row++) {
//         re = rl;
//         for (pixel = 1; pixel <= width; pixel++) {
//             its = fp_iterate(complex(re, im), EPSILON, 0);
//             r = cc[(its * 3) + 0];
//             g = cc[(its * 3) + 1];
//             b = cc[(its * 3) + 2];
//             if (write_pixel(bmp1, row, pixel, row_bytes, r, g, b) != 3) return;
//             re += d;
//         }
//         im += d;
//         printf("row %d of %d\r", row, height);
//     }
//     fclose(bmp1);
//     printf("\n\n");
// }