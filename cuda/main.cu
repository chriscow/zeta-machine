#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include "bitmap.hpp"
#include "zeta.hpp"

using sysclock = std::chrono::system_clock;
using sec = std::chrono::duration<double>;

extern "C" {
    //
    // size -   We always generate a square patch of data so width and height are
    //          the same.  Size is the size in one direction. The total data 
    //          generated is size * size so len(data) == size * size
    //
    // data -   Pre-allocated buffer of length size*size
    void generate(double minR, double maxR, double minI, double maxI, uint size, uint* data) {
        int procCount;
        int devId;
        hipGetDevice(&devId);
        hipDeviceGetAttribute(&procCount, hipDeviceAttributeMultiprocessorCount, devId);

        int threadsPerBlock;
        hipDeviceGetAttribute(&threadsPerBlock, hipDeviceAttributeMaxThreadsPerBlock, devId);

        int grids;
        grids = (size*size + threadsPerBlock - 1) / threadsPerBlock;

        uint* gpu_buffer;
        uint memsize = size * size * sizeof(uint);
        hipMalloc((void**)&gpu_buffer, memsize);

        complex min = complex(minR, minI);
        complex max = complex(maxR, maxI);

        calc_iterations<<<grids, threadsPerBlock>>>(min, max, size, gpu_buffer);

        hipDeviceSynchronize();
        hipMemcpy(data, gpu_buffer, memsize, hipMemcpyDeviceToHost);
        hipFree(gpu_buffer);

        write_image_(size, size, data);
    }
}

template<class T>
void prompt(const char *prompt, T *result) {
    std::cout << prompt << ":" << std::endl;
    std::cout << "> ";
    std::cin >> *result;
}

template<class T>
int menu(const char *title, T *options, int count) {
    for(;;) {
        std::cout << title << std::endl;
        for (int i = 0; i < count; i ++) {
            std::cout << i << ": " << options[i] << std::endl;
        }
        std::cout << "> ";
        int selected;
        std::cin  >> selected;
        if (selected < 0 || selected >= count) {
            selected = -1;
            std::cout << "No such option" << std::endl << std::endl;
        }
        else {
            return selected;
        }
    }
}

int main(void) {
    int procCount;
    int devId;
    hipGetDevice(&devId);
    hipDeviceGetAttribute(&procCount, hipDeviceAttributeMultiprocessorCount, devId);
    int threadsPerBlock;
    hipDeviceGetAttribute(&threadsPerBlock, hipDeviceAttributeMaxThreadsPerBlock, devId);

    std::cout << "proc count: " << procCount << std::endl;
    std::cout << "threads per block: " << threadsPerBlock << std::endl;

    Settings settings;

    uint size;
    prompt<double>("Real Min:", &settings.min.re);
    prompt<double>("Real Max:", &settings.max.re);
    prompt<double>("Imaginary Min:", &settings.min.im);
    prompt<double>("Imaginary Max:", &settings.max.im);
    prompt<uint>("Size (pixels wide):", &size);

    const auto before = sysclock::now();
    // write_image(settings);
    uint* data;
    data = (uint*)malloc(size*size * sizeof(uint));
    generate(
        settings.min.re,
        settings.max.re, 
        settings.min.im,
        settings.max.im, 
        size, data);

    free(data);
    const sec duration = sysclock::now() - before;
    std::cout << "It took " << duration.count() << "s" << std::endl;
    return 0;
}

